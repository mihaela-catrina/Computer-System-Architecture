#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>

#include "gpu_hashtable.hpp"

__device__ int hashFunc(int key, int capacity, int offset = 0) {
    if (offset == 0) return hash1(key, capacity);
    if (offset == 1) return hash2(key, capacity);
    if (offset == 2) return hash3(key, capacity);
    return hash3(key, capacity);
}

/**/
__global__ void cuckooInsert(int *keys, int *values, int numKeys, Bucket *table, int capacity, int *currentSize, int *updates) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numKeys) {
        int key = keys[tid];
        int value = values[tid];
        // create a new 64-bit value (key | value)
        Bucket newValue = (((static_cast<Bucket>(key) << INT_BITS)) | (static_cast<Bucket>(value)));
        
        // Compute all possible locations
	int idx[4];
	idx[0] = hashFunc(key, capacity, 0);
        idx[1] = hashFunc(key, capacity, 1);
        idx[2] = hashFunc(key, capacity, 2);
        
        Bucket entry;
        // verify if the key is already in the hashmap
        for (int i = 0; i < 3; ++i) {
            entry = static_cast<Bucket> (table[idx[i]]);
            Key k = (entry & HIGH) >> INT_BITS;
            // Update
            if (k == key) {
                newValue = atomicExch(&table[idx[i]], newValue);
                atomicAdd(updates, 1);
                return;
            }
        }
        // Othrewise try to insert the key in the table
        idx[0] = hashFunc(key, capacity, 0);
        // Max probe heuristic
        for (int i = 0; i < 7 *log2f(numKeys); ++i) {
	    // Exchange
            newValue = atomicExch(&table[idx[0]], newValue);
            if ((newValue & HIGH) >> INT_BITS == KEY_INVALID) {
              	 atomicAdd(currentSize, 1);
		 return;
            }
           
            key = (newValue & HIGH) >> INT_BITS;
            // Otherwise find a new location for the displaced item
            int last_loc = idx[0];
            idx[0] = hashFunc(key, capacity, 0);
            idx[1] = hashFunc(key, capacity, 1);
            idx[2] = hashFunc(key, capacity, 2);
            
            for (int i = 1; i >= 0; --i)
                idx[0] = (last_loc == idx[i] ? idx[i + 1] : idx[0]);

        }
    }
    return;
}

/**/
__global__ void cuckooGet(int *keys, int *values, int numKeys, Bucket *table, int capacity) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numKeys) {
        int key = keys[tid];

        // Compute all possible locations
        int loc[4];
        loc[0] = hashFunc(key, capacity, 0);
        loc[1] = hashFunc(key, capacity, 1);
        loc[2] = hashFunc(key, capacity, 2);

        Bucket entry;
        for (int i = 0; i < 3; ++i) {
            entry = table[loc[i]];
            Key k = (entry & HIGH) >> INT_BITS;
            // Get value
	    if (k == key) {
                int val = (entry & LOW);
                values[tid] = val;
                return;
            }
        } 
    }

    return;
}


/* INIT HASH
 */
GpuHashTable::GpuHashTable(int size) {

    capacity = 0;
    hipMallocManaged(&currentSize, 4);
    hipMallocManaged(&updates, 4);
    *updates = 0;
    *currentSize = 0;
}

/* DESTROY HASH
 */
GpuHashTable::~GpuHashTable() {
    hipFree(table);
    currentSize = 0;
    capacity = 0;
    table = NULL;
}

/* RESHAPE HASH
 */
void GpuHashTable::reshape(int numBucketsReshape) {
    Bucket *newTable = NULL;
    // free slots have <key, value> equal to 0
    hipMalloc((void **) &newTable, numBucketsReshape * sizeof(Bucket));
    hipMemset(newTable, KEY_INVALID, numBucketsReshape * sizeof(Bucket));
    if (table != NULL) {
        hipMemcpy(newTable, table, capacity * sizeof(Bucket), hipMemcpyDeviceToDevice);
        hipFree(table);
        capacity = numBucketsReshape;
    }
    capacity = numBucketsReshape;
    table = newTable;
}

/* INSERT BATCH
 */
bool GpuHashTable::insertBatch(int *keys, int *values, int numKeys) {

    hipMalloc(&deviceKeys, numKeys * sizeof(int));
    hipMalloc(&deviceValues, numKeys * sizeof(int));
    oldKeys = (int*)malloc(*currentSize * sizeof(int));
    oldValues = (int*)malloc(*currentSize * sizeof(int));
    hipMemcpy(deviceKeys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceValues, values, numKeys * sizeof(int), hipMemcpyHostToDevice);

    // Store the entries that are already present in the table (used in case of rehash)
    Bucket *tableValues = 0;
    tableValues = (Bucket *) malloc(capacity * sizeof(Bucket));
    hipMemcpy(tableValues, table, capacity * sizeof(Bucket), hipMemcpyDeviceToHost);
    int index = 0;
    for (int i = 0; i < capacity; i++) {
        if(tableValues[i] != KEY_INVALID) {
            oldValues[index] = (tableValues[i] & LOW);
            oldKeys[index] = (tableValues[i] & HIGH) >> INT_BITS;
            index++;
	}
        
    }
    oldSize = *currentSize;
    *updates = 0;
    free(tableValues);

    // Calculate reasonable block dimensions
    const int block_size = 64;
    int blocks_no = numKeys / block_size;
    if (numKeys % block_size)
        ++blocks_no;
    
    // Launch the kernel
    cuckooInsert <<< blocks_no, block_size >>> (deviceKeys, deviceValues, numKeys, table, capacity, currentSize, updates);
    hipDeviceSynchronize();
     // If all the values have not been successfully inserted
     if (*currentSize != oldSize + numKeys - *updates) {
        rehash(keys, values, numKeys);
    } else {
        // Copy values to the host part
        Bucket *hostValues = 0;
        hostValues = (Bucket *) malloc(capacity * sizeof(Bucket));
        hipMemcpy(hostValues, table, capacity * sizeof(Bucket), hipMemcpyDeviceToHost);
        // Free
        hipFree(deviceKeys);
        hipFree(deviceValues);
        free(oldKeys);
        free(oldValues);
        deviceKeys = 0;
        deviceValues = 0;
        oldKeys = 0;
        oldValues = 0;
    }

    return false;
}

void GpuHashTable::rehash(int *keys, int*values, int numKeys) {
    Bucket *newTable = NULL;
    Bucket *aux = 0;
    hipMalloc((void **) &newTable, 2 * capacity * sizeof(Bucket));
    hipMemset(newTable, KEY_INVALID, 2 * capacity * sizeof(Bucket));
    capacity *= 2;
    *currentSize = 0;

    aux = table;
    table = newTable;
    hipFree(aux);

    if (oldSize != 0) {
   	 insertBatch(oldKeys, oldValues, oldSize);
    }
    insertBatch(keys, values, numKeys);

}

/* GET BATCH
 */
int *GpuHashTable::getBatch(int *keys, int numKeys) {
    hipMalloc(&deviceKeys, numKeys * sizeof(int));
    hipMalloc(&deviceValues, numKeys * sizeof(int));
    hipMemcpy(deviceKeys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);

    const int block_size = 64;
    int blocks_no = numKeys / block_size;

    if (numKeys % block_size)
        ++blocks_no;
    cuckooGet <<< blocks_no, block_size >>> (deviceKeys, deviceValues, numKeys, table, capacity);
    hipDeviceSynchronize();
    int *hostValues = 0;
    hostValues = (int *) malloc(numKeys * sizeof(int));
    hipMemcpy(hostValues, deviceValues, numKeys * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(deviceKeys);
    hipFree(deviceValues);
    return hostValues;
}

/* GET LOAD FACTOR
 * num elements / hash total slots elements
 */
float GpuHashTable::loadFactor() {
    return *currentSize / (1.0 * capacity); // no larger than 1.0f = 100%
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#include "test_map.cpp"
