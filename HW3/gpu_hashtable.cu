#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>

#include "gpu_hashtable.hpp"

/* INIT HASH
 */
GpuHashTable::GpuHashTable(int size) {
	capacity = size;
    // free slots have <key, value> equal to 0
    CUDA_CALL(hipMalloc((void**)&table, capacity * sizeof(Bucket)));
    CUDA_CALL(hipMemset(table, 0, capacity * sizeof(Bucket)));
}

/* DESTROY HASH
 */
GpuHashTable::~GpuHashTable() {
    CUDA_CALL(hipFree(table));
}

/* RESHAPE HASH
 */
void GpuHashTable::reshape(int numBucketsReshape) {
}

/* INSERT BATCH
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	return false;
}

/* GET BATCH
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
	return NULL;
}

/* GET LOAD FACTOR
 * num elements / hash total slots elements
 */
float GpuHashTable::loadFactor() {
	return 0.f; // no larger than 1.0f = 100%
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#include "test_map.cpp"
