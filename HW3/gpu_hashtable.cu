#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>

#include "gpu_hashtable.hpp"

__device__ int hashFunc(int *hashConstants, int key, int capacity, int offset = 0) {
    // offset *= 2;
    // printf("Offset = %d\n", offset);
    // int a = hashConstants[19-offset];
    // int b = hashConstants[19-offset-1];
    // printf("a = %d, b = %d\n", a, b);
    // return ((a*key + b) % 4294967291U) % capacity;
    if (offset == 0) return hash1(key, capacity);
    if (offset == 1) return hash2(key, capacity);
    if (offset == 2) return hash3(key, capacity);
}

__global__ void cuckooInsert(int *keys, int *values, int numKeys, Bucket *table, int capacity, bool success) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numKeys) {
        int key = keys[tid];
        int value = values[tid];
        Bucket newValue = (((static_cast<Bucket>(key) << 32)) | (static_cast<Bucket>(value)));

        int idx[4];
        idx[0] = hashFunc(hashConstants, key, capacity, 0);
        for (int i = 0; i < 7 * log2f(numKeys); ++i) {
            newValue = atomicExch(&table[idx[0]], newValue);
            if ((newValue & 0xffffffff00000000) >> 32 == KEY_INVALID) {
                return;
            }
            key = (newValue & 0xffffffff00000000) >> 32;
            value = (newValue & 0x00000000ffffffff);
            // Otherwise find a new location for the displaced item
            int last_loc = idx[0];
            idx[0] = hashFunc(hashConstants, key, capacity, 0);
            idx[1] = hashFunc(hashConstants, key, capacity, 1);
            idx[2] = hashFunc(hashConstants, key, capacity, 2);
            //idx[3] = hashFunc(hashConstants, key, capacity, 3);

            for (int i = 1; i >= 0; --i)
                idx[0] = (last_loc == idx[i] ? idx[i + 1] : idx[0]);

        }
        /*
      if ((newValue & 0xffffffff00000000) >> 32 != KEY_INVALID)
  {	key = (newValue & 0xffffffff00000000)>>32;
      idx[0] = hashFunction(_hashConstants[0], key, capacity);
      auto slot = (unsigned long long int*)(table + (current_size + idx[0]));
      auto replaced = atomicCAS(slot, 0, newValue);
      if (replaced != 0) return;
      } */

        printf("tid %d: Insert (%u,%u) failed\n", tid, key, value);
        success = false;
    }
    return;
}

__global__ void cuckooGet(int *keys, int *values, int numKeys, Bucket *table, int capacity, bool success) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numKeys) {
        int key = keys[tid];

        // Compute all possible locations
        int idx[4];
        idx[0] = hashFunc(hashConstants, key, capacity, 0);
        idx[1] = hashFunc(hashConstants, key, capacity, 1);
        idx[2] = hashFunc(hashConstants, key, capacity, 2);
        // idx[3] = hashFunc(hashConstants, key, capacity, 3 );

        Bucket entry;
        for (int i = 0; i < 3; ++i) {
            entry = static_cast<Bucket> (table[idx[i] % capacity]);
            Key k = (entry & 0xffffffff00000000) >> 32;
            if (k == key) {
                int val = (entry & 0x00000000ffffffff);
                values[tid] = val;
                return;
            }
            if (k == KEY_INVALID)
                break;
        }

        // Should never fail except for invalid keys
        printf("Query for %u failed\n", key);
        success = false;
    }

    return;
}


/* INIT HASH
 */
GpuHashTable::GpuHashTable(int size) {

    capacity = 0;//size;
    currentSize = 0;
    // free slots have <key, value> equal to 0
//    hipMalloc((void **) &table, capacity * sizeof(Bucket));
//    hipMemset(table, KEY_INVALID, capacity * sizeof(Bucket));

}

/* DESTROY HASH
 */
GpuHashTable::~GpuHashTable() {
    hipFree(table);
    currentSize = 0;
    capacity = 0;
    table = NULL;
}

/* RESHAPE HASH
 */
void GpuHashTable::reshape(int numBucketsReshape) {
    Bucket *newTable = NULL;
    // free slots have <key, value> equal to 0
    hipMalloc((void **) &newTable, numBucketsReshape * sizeof(Bucket));
    hipMemset(newTable, KEY_INVALID, numBucketsReshape * sizeof(Bucket));
    if (table != NULL) {
        hipMemcpy(newTable, table, capacity * sizeof(Bucket), hipMemcpyDeviceToDevice);
        hipFree(table);
        capacity = numBucketsReshape;
    }
    capacity = numBucketsReshape;
    table = newTable;
}

/* INSERT BATCH
 */
bool GpuHashTable::insertBatch(int *keys, int *values, int numKeys) {

    hipMalloc(&deviceKeys, numKeys * sizeof(int));
    hipMalloc(&deviceValues, numKeys * sizeof(int));
    oldKeys = malloc(currentSize * sizeof(int));
    oldValues = malloc(currentSize * sizeof(int));
    hipMemcpy(deviceKeys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceValues, values, numKeys * sizeof(int), hipMemcpyHostToDevice);

    Bucket *tableValues = 0;

    tableValues = (Bucket *) malloc(capacity * sizeof(Bucket));
    hipMemcpy(tableValues, table, capacity * sizeof(Bucket), hipMemcpyDeviceToHost);
    int index = 0;
    for (int i = 0; i < capacity; i++) {
        if(tableValues[i] != KEY_INVALID) {
            oldValues[index] = (tableValues[i] & 0x00000000ffffffff);
            oldKeys[index] = (tableValues[i] & 0xffffffff00000000) >> 32;
        }
    }
    oldSize = currentSize;
    free(tableValues);

    currentSize += numKeys;


    printf("Insertttt\n");
    for (int i = 0; i < numKeys; i++) {
        printf("<key, value> : <%d, %d>\n", keys[i], values[i]);
    }
    const int block_size = 64;
    int blocks_no = numKeys / block_size;

    if (numKeys % block_size)
        ++blocks_no;
    bool success = true;
    // Launch the kernel
    cuckooInsert << < blocks_no, block_size >> > (deviceKeys, deviceValues, numKeys, table, capacity, &success);
    hipDeviceSynchronize();

    if (!success) {
        rehash(keys, values, numKeys);
        return false;
    }

    Bucket *hostValues = 0;

    hostValues = (Bucket *) malloc(capacity * sizeof(Bucket));
    hipMemcpy(hostValues, table, capacity * sizeof(Bucket), hipMemcpyDeviceToHost);
    printf("After insert in table:\n");
    for (int i = 0; i < capacity; i++) {
        printf("Key = %d -> ", (hostValues[i] & 0xffffffff00000000) >> 32);
        printf("Value = %d\n", (hostValues[i] & 0x00000000ffffffff));
    }

    hipFree(deviceKeys);
    hipFree(deviceValues);
    free(oldKeys);
    free(oldValues);
    deviceKeys = 0;
    deviceValues = 0;
    oldKeys = 0;
    oldValues = 0;
    hipDeviceSynchronize();
    return false;
}

void GpuHahTable::rehash(int *keys, int*values, int numKeys) {
    Bucket *newTable = NULL;
    BUcket *aux = 0;
    // free slots have <key, value> equal to 0
    hipMalloc((void **) &newTable, 2 * capacity * sizeof(Bucket));
    hipMemset(newTable, KEY_INVALID, 2 * capacity * sizeof(Bucket));
    capacity *= 2;
    aux = table;
    table = newTable;
    hipFree(aux);

    insertBatch(oldKeys, oldValues, oldSize);
    insertBatch(keys, values, numKeys);

}

/* GET BATCH
 */
int *GpuHashTable::getBatch(int *keys, int numKeys) {
    hipMalloc(&deviceKeys, numKeys * sizeof(int));
    hipMalloc(&deviceValues, numKeys * sizeof(int));
    hipMemcpy(deviceKeys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);

    bool success = true;
    const int block_size = 64;
    int blocks_no = numKeys / block_size;

    if (numKeys % block_size)
        ++blocks_no;
    cuckooGet << < blocks_no, block_size >> > (deviceKeys, deviceValues, numKeys, table, capacity, success);
    hipDeviceSynchronize();
    int *hostValues = 0;
    hostValues = (int *) malloc(numKeys * sizeof(int));
    hipMemcpy(hostValues, deviceValues, numKeys * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(deviceKeys);
    hipFree(deviceValues);
    return hostValues;
}

/* GET LOAD FACTOR
 * num elements / hash total slots elements
 */
float GpuHashTable::loadFactor() {
    return currentSize / (1.0 * capacity); // no larger than 1.0f = 100%
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#include "test_map.cpp"
