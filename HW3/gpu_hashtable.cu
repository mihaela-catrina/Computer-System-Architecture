#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>

#include "gpu_hashtable.hpp"

__device__ int hash(int key, int offset = 0 )
{
    offset *= 2;
    uint64_t a = primeList[MAX_HASH_PARAM-offset];
    uint64_t b = primeList[MAX_HASH_PARAM-offset-1];
    return (a*key + b) % 4294967291U;
}

__global__ void cuckooInsert( int* keys, int* values, int numKeys, Bucket *table, int capacity)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numKeys) {
        int key = keys[idx];
        int value = values[idx];
        Bucket newValue = {key, value};
        int oldHashIdx;

        hashIdx = hash(key, 0) % capacity;
        for (int i = 0; i < MAX_VER; ++i) {
            newValue = atomicExch(table[hashIdx], newValue);
            if (newValue.key == KEY_INVALID)
                return;
            oldHashIdx = hashIdx;
            for (int j = 0; j < MAX_VER; ++j) {
                hashIdx = hash(key, j) % capacity;
                if (hashIdx != oldHashIdx)
                    break;
            }
        }

        printf("tid %d: Insert (%u,%u) failed\n", idx, key, value);
    }
    return;
}



/* INIT HASH
 */
GpuHashTable::GpuHashTable(int size) {

    capacity = size;
    // free slots have <key, value> equal to 0
    hipMalloc((void **) &table, capacity * sizeof(Bucket));
    hipMemset(table, KEY_INVALID, capacity * sizeof(Bucket));
}

/* DESTROY HASH
 */
GpuHashTable::~GpuHashTable() {
    hipFree(table);
    currentSize = 0;
    capacity = 0;
    table = NULL;
}

/* RESHAPE HASH
 */
void GpuHashTable::reshape(int numBucketsReshape) {

    Bucket *newTable = NULL;
    // free slots have <key, value> equal to 0
    hipMalloc((void **) &newTable, numBucketsReshape * sizeof(Bucket));
    hipMemset(newTable, KEY_INVALID, numBucketsReshape * sizeof(Bucket));
    if (table != NULL) {
        hipMemcpy(newTable, table, capacity * sizeof(Bucket), hipMemcpyDeviceToDevice);
        hipFree(table);
        capacity = numBucketsReshape;
    }
}

/* INSERT BATCH
 */
bool GpuHashTable::insertBatch(int *keys, int *values, int numKeys) {

    currentSize += numKeys;
    hipMalloc(&deviceKeys, numKeys * sizeof(int));
    hipMalloc(&deviceValues, numKeys * sizeof(int));
    hipMemcpy(deviceKeys, keys, numKeys, hipMemcpyHostToDevice);
    hipMemcpy(deviceValues, values, numKeys, hipMemcpyHostToDevice);

    const int block_size = 64;
    int blocks_no = numKeys / block_size;

    if (numKeys % block_size)
        ++blocks_no;

    // Launch the kernel
    cuckooInsert <<< blocks_no, block_size >>> (deviceKeys, numKeys, deviceValues, table, capacity);

    hipDeviceSynchronize();
    return false;
}

/* GET BATCH
 */
int *GpuHashTable::getBatch(int *keys, int numKeys) {
    return NULL;
}

/* GET LOAD FACTOR
 * num elements / hash total slots elements
 */
float GpuHashTable::loadFactor() {
    return currentSize / 1.0 * capacity; // no larger than 1.0f = 100%
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#include "test_map.cpp"
